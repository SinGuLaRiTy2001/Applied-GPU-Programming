#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define DataType double

static int ThreadNum = 256;
static const int numStreams = 4; // Number of CUDA streams

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        out[idx] = in1[idx] + in2[idx];
    }
}

double getTime() {
    struct timespec time_now;
    clock_gettime(CLOCK_REALTIME, &time_now);
    double RunTime = (double)time_now.tv_sec + (double)time_now.tv_nsec / 1.e9;
    return RunTime;
}

int main(int argc, char **argv) {
    int inputLength;
    int S_seg;
    DataType *hostInput1;
    DataType *hostInput2;
    DataType *hostOutput;
    DataType *resultRef;
    DataType *deviceInput1;
    DataType *deviceInput2;
    DataType *deviceOutput;

    double startTime, endTime;

    inputLength = atoi(argv[1]);
    S_seg = atoi(argv[2]); // Segment size
    printf("The input length is %d\n", inputLength);
    printf("Segment size: %d\n", S_seg);

    hostInput1 = (DataType *) malloc(inputLength * sizeof(DataType));
    hostInput2 = (DataType *) malloc(inputLength * sizeof(DataType));
    hostOutput = (DataType *) malloc(inputLength * sizeof(DataType));
    resultRef = (DataType *) malloc(inputLength * sizeof(DataType));

    for (int i = 0; i < inputLength; i++) {
        hostInput1[i] = rand() / (DataType) (RAND_MAX + 1.0);
        hostInput2[i] = rand() / (DataType) (RAND_MAX + 1.0);
        resultRef[i] = hostInput1[i] + hostInput2[i];
    }

    hipMalloc(&deviceInput1, inputLength * sizeof(DataType));
    hipMalloc(&deviceInput2, inputLength * sizeof(DataType));
    hipMalloc(&deviceOutput, inputLength * sizeof(DataType));

    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; i++) {
        hipStreamCreate(&streams[i]);
    }

    startTime = getTime();
    for (int i = 0; i < inputLength; i += S_seg) {
        int segSize = min(S_seg, inputLength - i);
        int streamIdx = i / S_seg % numStreams;

        hipMemcpyAsync(deviceInput1 + i, hostInput1 + i, segSize * sizeof(DataType), hipMemcpyHostToDevice, streams[streamIdx]);
        hipMemcpyAsync(deviceInput2 + i, hostInput2 + i, segSize * sizeof(DataType), hipMemcpyHostToDevice, streams[streamIdx]);

        int Block_dim = ThreadNum;
        int Grid_dim = segSize / ThreadNum + (segSize % ThreadNum != 0);
        vecAdd<<<Grid_dim, Block_dim, 0, streams[streamIdx]>>>(deviceInput1 + i, deviceInput2 + i, deviceOutput + i, segSize);

        hipMemcpyAsync(hostOutput + i, deviceOutput + i, segSize * sizeof(DataType), hipMemcpyDeviceToHost, streams[streamIdx]);
    }

    for (int i = 0; i < numStreams; i++) {
        hipStreamSynchronize(streams[i]);
    }
    endTime = getTime();
    printf("Total asynchronous operation time: %.6lf ms.\n", (endTime - startTime) * 1000);

    bool flag = true;
    for (unsigned int i = 0; i < inputLength; i++) {
        if (fabs(hostOutput[i] - resultRef[i]) > 1e-10) {
            flag = false;
            break;
        }
    }

    if (flag)
        printf("Result match reference.\n");
    else
        printf("Result is different from reference.\n");

    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);
    free(resultRef);

    for (int i = 0; i < numStreams; i++) {
        hipStreamDestroy(streams[i]);
    }

    return 0;
}
